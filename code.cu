#include "hip/hip_runtime.h"
%%cu
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>
#include <iomanip>

using namespace std;

const int MAX_NAME_LENGTH = 100;
const int MAX_RESULT_STRING_LENGTH = 200;

struct Product {
    char Name[MAX_NAME_LENGTH];
    int Id;
    double Cost;
};

struct ProductResult {
    char Name[MAX_NAME_LENGTH];
    int Id;
    double Cost;
    char ComputedData[MAX_RESULT_STRING_LENGTH];
};

//Paleidzia ir vykdo GPU
__device__ void manualStrcpy(char* dest, const char* src) {
    int i = 0;
    while (src[i] != '\0') {
        dest[i] = src[i];
        i++;
    }
    dest[i] = '\0'; // Null-terminate the string
}

__device__ void floatToStr(double value, char* str) {
    // Basic conversion from a float to a string
    int intPart = static_cast<int>(value);
    double fracPart = value - static_cast<double>(intPart);
    int fracPartInt = static_cast<int>(fracPart * 1000000); // 6 decimal places

    // Convert integer part
    int i = 0;
    if (intPart == 0) {
        str[i++] = '0';
    } else {
        char temp[20];
        int j = 0;
        while (intPart != 0) {
            temp[j++] = '0' + (intPart % 10);
            intPart /= 10;
        }
        for (j = j - 1; j >= 0; j--) {
            str[i++] = temp[j];
        }
    }

    str[i++] = '.'; // Decimal point

    // Convert fractional part
    if (fracPartInt == 0) {
        str[i++] = '0';
    } else {
        char temp[20];
        int j = 0;
        while (fracPartInt != 0) {
            temp[j++] = '0' + (fracPartInt % 10);
            fracPartInt /= 10;
        }
        for (j = j - 1; j >= 0; j--) {
            str[i++] = temp[j];
        }
    }

    str[i] = '\0'; // Null-terminate the string
}

__global__ void CalculationsKernel(const Product* products, ProductResult* results, int* validCount, int numProducts) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Global thread index
    int stride = blockDim.x * gridDim.x; // Total number of threads

    for (int i = idx; i < numProducts; i += stride) { // Loop through all products
        Product product = products[i];
        ProductResult result;
        double asciiSum = 0;

        // Calculate ASCII sum of the product name
        for (int j = 0; j < MAX_NAME_LENGTH && product.Name[j] != '\0'; ++j) {
            asciiSum += static_cast<int>(product.Name[j]);
        }

        // Calculate product.Cost to the power of 2
        double costPowerTwo = pow(product.Cost, 2);

        // Calculation combining ASCII sum and cost to the power of 2
        double combinedValue = asciiSum + costPowerTwo;

        // Check and add to results if combinedValue is above a certain threshold
        double threshold = 4000.0;
        if (combinedValue > threshold) {
            int insertIdx = atomicAdd(validCount, 1);

            // Copying the product data to result
            manualStrcpy(result.Name, product.Name);
            result.Id = product.Id;
            result.Cost = product.Cost;

            // Storing combinedValue as a string in ComputedData
            floatToStr(combinedValue, result.ComputedData);

            results[insertIdx] = result;
        }
    }
}

void RunCalculationsOnGPU(Product* products, ProductResult* results, int* validCount, int numProducts) {
    // Pointer to memory on the GPU
    Product* d_products;
    ProductResult* d_results;
    int* d_validCount;

    // Allocate memory on the GPU
    hipMalloc((void**)&d_products, numProducts * sizeof(Product));
    hipMalloc((void**)&d_results, numProducts * sizeof(ProductResult));
    hipMalloc((void**)&d_validCount, sizeof(int));
    hipMemset(d_validCount, 0, sizeof(int));

    // Copy data from CPU to GPU
    hipMemcpy(d_products, products, numProducts * sizeof(Product), hipMemcpyHostToDevice);

    int threadsPerBlock = 32;
    int numBlocks = (numProducts + threadsPerBlock - 1) / threadsPerBlock;
    numBlocks = max(2, numBlocks); // At least two blocks

    // Run the kernel
    CalculationsKernel<<<numBlocks, threadsPerBlock>>>(d_products, d_results, d_validCount, numProducts);

    // Copy results from GPU to CPU
    hipMemcpy(results, d_results, numProducts * sizeof(ProductResult), hipMemcpyDeviceToHost);
    hipMemcpy(validCount, d_validCount, sizeof(int), hipMemcpyDeviceToHost);

    // Free memory on the GPU
    hipFree(d_products);
    hipFree(d_results);
    hipFree(d_validCount);
}

void ReadProducts(const string& dataFile, Product products[], int& index) {
    ifstream file(dataFile);

    if (!file.is_open()) {
        cout << "Failed to open the file: " << dataFile << endl;
        return;
    }

    index = 0;
    char semicolon;
    while (file >> ws && !file.eof()) {
        file.getline(products[index].Name, MAX_NAME_LENGTH, ';'); // Read until the semicolon
        file >> products[index].Id >> semicolon >> products[index].Cost; // Read the rest of the line
        file.ignore(numeric_limits<streamsize>::max(), '\n'); // Ignore the rest of the line
        index++;
    }

    file.close();
}

void PrintResults(const ProductResult results[], int validCount, const string& fileName) {
    ofstream out(fileName);

    out << "----------------------------------------------------------------------------------------" << endl;
    out << "| " << setw(33) << "Produktas" << " | " << setw(6) << "Id" << " | " << setw(6) << "Kaina" << " |" << setw(30) << "           Apskaičiuota reikšmė" << " |" << endl;
    out << "----------------------------------------------------------------------------------------" << endl;

    for (int i = 0; i < validCount; i++) {
        out << "| " << setw(33) << results[i].Name
            << " | " << setw(6) << results[i].Id
            << " | " << setw(6) << results[i].Cost
            << " | " << setw(30) << results[i].ComputedData << " |" << endl;
    }

    out << "----------------------------------------------------------------------------------------" << endl;
    out.close();
}

int main() {
    string inputFile1 = "./data/IFF-1-5_AndziulisJ_L1_dat_1.txt";
    string inputFile2 = "./data/IFF-1-5_AndziulisJ_L1_dat_2.txt";
    string inputFile3 = "./data/IFF-1-5_AndziulisJ_L1_dat_3.txt";
    string outputFile = "./output.txt"; // Replace with output file name
    int numProducts = 1000;
    Product* products = new Product[numProducts];
    ProductResult* results = new ProductResult[numProducts];
    int validCount = 0;

    ReadProducts(inputFile2, products, numProducts); // Replace with input file
    RunCalculationsOnGPU(products, results, &validCount, numProducts);
    PrintResults(results, validCount, outputFile);

    delete[] products;
    delete[] results;

    return 0;
}